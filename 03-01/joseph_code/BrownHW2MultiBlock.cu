/*
Joseph Brown
Homework2

Problem1
Allows arbitrarily large vectors to be processed by using multiple blocks.
*/


#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include <math.h>

int N = 100;
	//Sets N to the default value of 100.

float *A_CPU, *B_CPU, *C_CPU;

float *A_GPU, *B_GPU, *C_GPU;

dim3 dimBlock;	//

void AllocateMemory(){
	hipMalloc((void**)&A_GPU, N*sizeof(float));
	hipMalloc((void**)&B_GPU, N*sizeof(float));
	hipMalloc((void**)&C_GPU, N*sizeof(float));
	
	A_CPU = (float*)malloc(N*sizeof(float));
	B_CPU = (float*)malloc(N*sizeof(float));
	C_CPU = (float*)malloc(N*sizeof(float));
} // */	//Saves the appropriate memory chunks for later use.
	//References the globally defined variables.

void Initialize(){
	for(int i = 0; i < N; i++){
		A_CPU[i] = (float)i;
		B_CPU[i] = (float)i;
	}	//Sets these arrays to the values 1..N.		
} // */

void CleanUp(float *A_CPU, float *B_CPU, float *C_CPU, 
	     float *A_GPU, float *B_GPU, float *C_GPU){
	free(A_CPU);
	free(B_CPU);
	free(C_CPU);

	hipFree(A_GPU);
	hipFree(B_GPU);
	hipFree(C_GPU);
} // */	//Frees the memory for the three relevant global variables.

/*void VectorAddition(float *A, float *B, float *C, int n){
	cudaMemcpy(A_GPU, A_CPU, N*sizeof(float), cudaMemcpyHostToDevice);
	cudaMemcpy(B_GPU, B_CPU, N*sizeof(float), cudaMemcpyHostToDevice);
	add<<<N,1>>>(A_GPU, B_GPU, C_GPU)
	cudaMemcpy(C_CPU, C_GPU, N*sizeof(float), cudaMemcpyDeviceToHost);
} // */	//Takes the component-wise sum of the first n 
	// values of two vectors, A and B, and stores them in the
	// corresponding values of a third vector, C.

__global__ void Addition(float *A, float *B, float *C, int n){
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if(tid < n){
		C[bid*1024 + tid] = A[bid*1024 + tid] + B[bid*1024 +tid]; 
	}
}


int main(int argc, char *argv[]){
	//I would like this program to accept command line
	// arguments.  Simply run "./VectorAdditionCPU.cu #"
	// to run the same program with a different parameter.

	timeval start, end;
		//Declares two objects of type timeval.

	if(argc == 2){
		char *ptr;
		N = strtol(argv[1], &ptr, 10);
	}
	else if(argc > 2){
		printf("One or zero arguments expected.");
		return(1);
	}

	AllocateMemory();
	Initialize();
	gettimeofday(&start,NULL);
	int numblocks = (N-1)/1024 + 1;
			//Required number of blocks of size 1024,
			// plus one for the leftovers.  -NST

	hipMemcpy(A_GPU, A_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_GPU, B_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	Addition<<<numblocks,1024>>>(A_GPU, B_GPU, C_GPU, N);
	hipMemcpy(C_CPU, C_GPU, N*sizeof(float), hipMemcpyDeviceToHost);
	//VectorAddition(A_CPU, B_CPU, C_CPU, N);
	gettimeofday(&end, NULL);
	float time = (end.tv_sec*1000000 + end.tv_usec*1) - 
		     (start.tv_sec*1000000 + start.tv_usec*1);
		     //tv_sec is in seconds, while tv_usec
		     // is in microseconds, so they need to
		     // be scaled appropriately.
		     //Then, it's a matter of subtracting
		     // the value of the start from the
		     // value of the end.
	printf("CPU Time in milliseconds= %.10f\n", (time/1000.0));
	printf("Blocks used= %d\n", numblocks);
	for(int i = 0; i < 5; i++){
		printf("A[%d] = %.5f   B[%d] = %.5f   C[%d] = %.5f\n",
			  i,    A_CPU[i], i,    B_CPU[i], i,    C_CPU[i]);
	} // */
	/*for(int i = 5; i < N-1; i++){
		printf("A[%d] = %.5f   B[%d] = %.5f   C[%d] = %.5f\n",
			  i,   A_CPU[i], i,    B_CPU[i], i,    C_CPU[i]);
	}// */
	printf("...\n");// */
	printf("A[%d] = %.5f   B[%d] = %.5f   C[%d] = %.5f\n",
		  N-1,  A_CPU[N-1],N-1, B_CPU[N-1],N-1, C_CPU[N-1]);
	
	CleanUp(A_CPU, B_CPU, C_CPU,
		A_GPU, B_GPU, C_GPU);

	return(0);
}

//Output for:
//ccudaclass2016@lannister:/media/storage/CUDAClasses/CUDACLASS2017/JosephBrown/Homework2$ ./temp1 46565
// where temp1 is the compiled version of this script run on lannister with arg 46565.
/*
CPU Time in milliseconds= 0.4740000000
Blocks used= 46
A[0] = 0.00000   B[0] = 0.00000   C[0] = 0.00000
A[1] = 1.00000   B[1] = 1.00000   C[1] = 2.00000
A[2] = 2.00000   B[2] = 2.00000   C[2] = 4.00000
A[3] = 3.00000   B[3] = 3.00000   C[3] = 6.00000
A[4] = 4.00000   B[4] = 4.00000   C[4] = 8.00000
...
A[46564] = 46564.00000   B[46564] = 46564.00000   C[46564] = 93128.00000
*/

